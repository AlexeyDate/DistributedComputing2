#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define N 100

__global__ void calcGauss(float *A, float *B) {
    int col = threadIdx.x;

    for (int k = 0; k < N - 1; ++k) {
        if (col > k) {
            float factor = A[N * (col) + k] / A[N * k + k];
            for (int j = k; j < N; ++j) {
                A[N * col + j] -= factor * A[N * k + j];
            }
            B[col] -= factor * B[k];
        }
        __syncthreads();
    }

    if (col == N - 1) {
        B[N - 1] /= A[N * (N - 1) + (N - 1)];
        for (int i = N - 2; i >= 0; --i) {
            float sum = 0.0;
            for (int j = i + 1; j < N; ++j) {
                sum += A[N * i + j] * B[j];
            }
            B[i] = (B[i] - sum) / A[N * i + i];
        }
    }
}

int main() {
    float A[N * N], A1[N*N];
    float B[N], B1[N], Res[N];

    srand(time(0));
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            A[N * i + j] = rand() % (N * 100);
            A1[N * i + j] = A[N * i + j];
        }
        B[i] = rand() % (N * 100);
        B1[i] = B[i];
    }

    if (N < 30)
    {
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                std::cout << A[N * i + j] << "\t";
            }
            std::cout << "| " << B1[i] << std::endl;
        }
    }

    float *device_A, *device_B;
    hipMalloc((void**)&device_A, N * N * sizeof(float));
    hipMalloc((void**)&device_B, N * sizeof(float));

    hipMemcpy(device_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    calcGauss<<<1, N>>>(device_A, device_B);

    hipMemcpy(A, device_A, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(B, device_B, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(device_A);
    hipFree(device_B);

    float sum = 0;
    for (int i = 0; i < N; ++i){
        float res = 0;
        for (int j = 0; j < N; ++j)
        {
             res += A1[i*N+j] * B[j];  
        }   
        Res[i] = res;
        sum += abs(res - B1[i]);
    }

    if (N < 50)
    {
        std::cout << "\nСheck:" << std::endl;
        for (int i = 0; i < N; ++i)
        {
            std::cout << "Expected result: " << B1[i] << ", Calculated result: " << Res[i] << std::endl << ", Error: " << Res[i] - B1[i] << std::endl;
        }

        printf("Transformed vector B:\n");
        for (int i = 0; i < N; ++i) {
            printf("X%d = %.2f\n", i, B[i]);
        }
    }
    else
      printf("Mean error: %.5f", (float)sum / N);

    return 0;
}